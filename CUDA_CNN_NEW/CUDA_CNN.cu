#include "hip/hip_runtime.h"


/*

Yuzhe Shang, Project 6, blur the dataset
ECE 350/450 Accelerated Computing for Deep Learning
17:00/May 08/2020
References:
https://github.com/csuldw/MachineLearning/blob/master/utils/data_util.py (Read and write the MNIST)
http://www.voidcn.com/article/p-rzninoud-bbs.html (Drag data from MNIST using C++)
https://tschmidt23.github.io/cse599i/CSE%20599%20I%20Accelerated%20Computing%20-%20Programming%20GPUs%20Lecture%203.pdf (Image Blur as 2D kernal)
https://www.cnblogs.com/hjj-fighting/p/10429178.html (How to write the dataset to a file using C++)
timer.h and the timing method used in 01-nbody.cu.
01-nbody-gpu.cu
https://blog.csdn.net/u010579901/article/details/78852879


*/

/*First part is to read the MNIST dataset*/

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include "timer.h"//All the head files below are from nbody-gpu.cu
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#define DimBlock 256 //Define the Dimblock
#define n 28 // Define the thread_per_block
using namespace std;
 
int ReverseInt(int i)  //Integer reverse, change the data into binary
{
	unsigned char ch1, ch2, ch3, ch4;
	ch1 = i & 255;
	ch2 = (i >> 8) & 255;
	ch3 = (i >> 16) & 255;
	ch4 = (i >> 24) & 255;
	return((int)ch1 << 24) + ((int)ch2 << 16) + ((int)ch3 << 8) + ch4;
}
 

 
void read_Mnist_Images(vector<vector <double> > &images) //Read dataset
{
	ifstream file("t10k-images.idx3-ubyte", ios::binary);  //Read the MNIST in binary
	if (file.is_open())
	{
		int magic_number = 0;
		int number_of_images = 0;
		int n_rows = 0;
		int n_cols = 0;
	
		file.read((char*)&magic_number, sizeof(magic_number));
		file.read((char*)&number_of_images, sizeof(number_of_images));
		file.read((char*)&n_rows, sizeof(n_rows));
		file.read((char*)&n_cols, sizeof(n_cols));
		magic_number = ReverseInt(magic_number);
		number_of_images = ReverseInt(number_of_images);
		n_rows = ReverseInt(n_rows);
		n_cols = ReverseInt(n_cols);

 		cout << "magic number = " << magic_number << endl; // show magic number
		cout << "number of images = " << number_of_images << endl; // show inmage numbers
		cout << "rows = " << n_rows << endl; //show number of rows
		cout << "cols = " << n_cols << endl; //show number of cols


		for (int i = 0; i < 10000; i++)   //number_of_images, read 10000 images
		{
			vector<double>tp;
			for (int r = 0; r < n_rows; r++)
			{
				for (int c = 0; c < n_cols; c++)
				{
					unsigned char image = 1;
					file.read((char*)&image, sizeof(image));
					tp.push_back(image);
				}
			}
			images.push_back(tp);
		}
	}
}


/*GPU kernel code, initiated by CPU, cannot be called by other kernel*/
__global__
void blurkernel (unsigned char *in, unsigned char *out, int w, int h)
{

int row = blockIdx.x * blockDim.x + threadIdx.x;
int col = blockIdx.y * blockDim.y + threadIdx.y;

if (col < h && row < w )
{

int pixVal = 0;
int pixels = 0; 
int BLUR_SIZE = 1; // Define the blur size as 1
for (int blurcol = -BLUR_SIZE; blurcol < BLUR_SIZE+1; ++blurcol){
 for (int blurrow = -BLUR_SIZE; blurrow < BLUR_SIZE+1; ++blurrow)
{
 int currow = row + blurrow;
 int curcol = col + blurcol;

  if (currow > -1 && currow < w && curcol > -1 && curcol < h)
{
 pixVal += in[curcol * w + currow];

 pixels++;   //Keep track of number of pixels in the accumulated total
     }
   }
 }

out[col * w + row] = (unsigned char)(pixVal / pixels);  //Write our new pixel value out

   }
                                           
}

/*use main function*/
int main()
{	
	vector< vector<double> > images;
	read_Mnist_Images(images);   //Read images


 ofstream outFile("train5.idx3-ubyte",  ios::binary);  //use ofstream to create a new file named train5.idx3-ubyte
   
 for (int i = 0; i < images.size(); i++)
	{
		for (int j = 0; j < images[0].size(); j++)   //for (auto iter = labels.begin(); iter != labels.end(); iter++)
		{ 
        	
		}
	}



/*Image Blur 10 times as a 2D Kernel*/

/*unsigned char in, unsigned char out, int w, int h*/


unsigned char *in; //Define the size of char in and out
unsigned char *out;
int w = images.size(); //Define the size of height and width
int h = images[0].size();


int size = 250000 *n * n * sizeof (unsigned char);// Changed the float to char, increase the memory
 
  hipMallocManaged (&out, size);
  hipMallocManaged (&in, size);

for(int col =0;col < w ;col++)
{
for(int row =0;row < h ;row++)
{
in[col*w+row] = images[col][row];                                      
out[col*w+row] = images[col][row]; 
}
}

/*code below are from the 01-nbody-gpu.cu*/
const int nIters = 1;//Blur one time
double totalTime = 0.0;

 for (int iter = 1; iter <= nIters; iter++) {
  StartTimer();


/*Blur the image*/
for(int col =0;col < w ;col++)
{
for(int row =0;row < h ;row++)
{

//in [col*w+ row] =  out [col*w+ row]    ;                         



}
}

  dim3 Dimblock (28, 28, 1); // 2D, n*n*1 
  dim3 Dimgrid ((n - 1 / Dimblock.x) + 1, (n - 1/ Dimblock.y) + 1, 1); //Total 3 dimension, I used 2 dims

 blurkernel <<<Dimblock, Dimblock>>> (in, out, 28, 28); //Write all parameters in the kernal

hipDeviceSynchronize();// synchronized, Waiting for GPU kernel execution to end
    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;

}


for(int col =0;col < w ;col++)
{
for(int row =0;row < h ;row++)
{

                               
outFile.write((char*)&out[col*w+ row], sizeof(out[col*w+ row]));


}
}

cout <<totalTime  << " ";
	return 0;
 

}
